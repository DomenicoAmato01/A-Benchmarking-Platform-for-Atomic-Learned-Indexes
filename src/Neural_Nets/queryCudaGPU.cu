#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <hiprand.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <algorithm>

#define RES 1e9


/*
* Argument Parser
*/
char* getCmdOption(char ** begin, char ** end, const std::string & option)
{
    char ** itr = std::find(begin, end, option);
    if (itr != end && ++itr != end)
    {
        return *itr;
    }
    return 0;
}

bool cmdOptionExists(char** begin, char** end, const std::string& option)
{
    return std::find(begin, end, option) != end;
}

/*
* Matrix Utility
*/
long readMatrix(const char* fn, float **data, int m){

    FILE* fp;
    long num = 0;
    fp = fopen( fn, "r" );
    if(fp == NULL) std::cout << "Errore apertura file: " << errno << std::endl;

    std::cout <<"Leggo il numero di righe" << std::endl;
    fscanf(fp, "%ld\n", &num);

    std::cout << "Alloco " << num << std::endl;
    *data = (float*)malloc(sizeof(float)*(num*m));
    std::cout << "Leggo i valori" << std::endl;
    int aus = 0;
    for(int i = 0; i<num; i++){
        for(int j = 0; j<m;j++){
            fscanf(fp, "%1d", &aus);
            data[0][j * num + i] = (float)aus;
            //std::cout << data[0][j * num + i];
        }
        //std::cout << std::endl;
        //fgetc(fp);
    }
    //printf("A[]")
    fclose(fp);
    std::cout << "Ritorno numero di righe" << std::endl;
    return num;
}

void readNNParams(const char* fn, float **data, int m){

    FILE* fp;
    fp = fopen( fn, "r" );
    if(fp == NULL) std::cout << "Errore apertura file: " << errno << std::endl;
    //printf("Leggo Parametri");
    std::cout << "Alloco Parametri" << std::endl;
    *data = (float*)malloc(sizeof(float)*(m));
    std::cout << "Leggo i valori" << std::endl;
    float aus = 0.0;
    for(int j = 0; j<m;j++){
        fscanf(fp, "%f\n", &aus);
        data[0][j] = aus;
        //std::cout << data[0][j] << std::endl;
    }


    fclose(fp);
}

/*
* Cuda Operations
*/

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(hipblasHandle_t &handle, const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;
	//Do the actual multiplication
	//hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha,
		                          A, HIP_R_32F, lda,
		                          B, HIP_R_32F, ldb,
								  beta, C, HIP_R_32F, ldc, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
}
/*
void tpu_blas_mmul(hipblasHandle_t &handle, const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	//Do the actual multiplication
	//hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha,
	                          A, HIP_R_32F, lda,
	                          B, HIP_R_32F, ldb,
	                          beta, C, HIP_R_32F, ldc, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}
*/
__global__ void k_relu(float *a, float *c)
{
    int idx = blockIdx.x;
    c[idx] = a[idx]>0 ? a[idx] : a[idx]*0.05;
}

__global__ void k_scalar_matrix_sum(float *a, float *b, float *c)
{
    int idx = blockIdx.x;
    c[idx] = a[idx] + *b;
}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

	for(int i = 0; i < nr_rows_A; ++i){
		for(int j = 0; j < nr_cols_A; ++j){
		   std::cout << A[j * nr_rows_A + i] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

void print_matrix_float(const float *A, int nr_rows_A, int nr_cols_A) {

	for(int i = 0; i < nr_rows_A; ++i){
		for(int j = 0; j < nr_cols_A; ++j){
		   std::cout << A[j * nr_rows_A + i] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

/*
* Prediction Function
*/

void prediction(const char *fn,const char *wfn1,const char *bfn1, const char *output){
	
	//Name of data file
	//const char *fn="Resource/lognormal/Query/file3lognormalQuery10_bin_c.dat";
	//const char *wfn1="Resource/lognormal/weights_dat/weights_nn1_file3_c.dat";
	//const char *bfn1="Resource/lognormal/weights_dat/bias_nn1_file3_c.dat";


	int nr_rows_A, nr_cols_A,
	nr_rows_W1, nr_cols_W1,
	nr_rows_C1, nr_cols_C1,
	nr_bias1;

	nr_bias1 = nr_cols_W1 = 1;

	float *h_A;
	float *h_W1;
	float *h_bias1;

	struct timespec requestStart, requestEnd;
	double timer;
	double copy_acc=0, mult_acc=0, bias_acc=0, relu_acc=0;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasStatus_t cublasStat =  hipblasCreate(&handle);

	FILE* out_fp;
    out_fp = fopen( output, "a" );
    if(out_fp == NULL) std::cout << "Errore apertura file Output: " << errno << std::endl;

	nr_cols_A = nr_rows_W1 = 64;
	nr_rows_A = (int)readMatrix(fn, &h_A, nr_cols_A);

	nr_rows_C1 = nr_rows_A;
	nr_cols_C1 = nr_cols_W1;
	readNNParams(wfn1, &h_W1, nr_cols_A*nr_cols_W1);
	readNNParams(bfn1, &h_bias1, nr_bias1);

/*
	std::cout << "A =" << std::endl;
	print_matrix(h_A, nr_rows_A, nr_cols_A);

	std::cout << "W =" << std::endl;
	print_matrix(h_W1, nr_rows_W1, nr_cols_W1);
*/
	float *h_C1 = (float *)malloc(nr_rows_C1 * nr_cols_C1 * sizeof(float));

	// Allocate 3 arrays on GPU
	float *d_A, *d_W1, *d_bias1;
	float *d_C1;
	hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
	hipMalloc(&d_W1,nr_rows_W1 * nr_cols_W1 * sizeof(float));
	hipMalloc(&d_bias1,nr_bias1 * sizeof(float));
	hipMalloc(&d_C1,nr_rows_C1 * nr_cols_C1 * sizeof(float));


	clock_gettime(CLOCK_REALTIME, &requestStart);
	hipMemcpy(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_W1,h_W1,nr_rows_W1 * nr_cols_W1 * sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_bias1,h_bias1,nr_bias1 * sizeof(float),hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	/*clock_gettime(CLOCK_REALTIME, &requestEnd);
	timer = (double)( requestEnd.tv_sec - requestStart.tv_sec )
	+ (double)( requestEnd.tv_nsec - requestStart.tv_nsec )
	/ RES;
	std::cout << "Time Copy Input TPU: " << timer << std::endl;
	copy_acc += timer;
*/
	// Set the math mode to allow cuBLAS to use Tensor Cores:
	cublasStat = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
	//clock_gettime(CLOCK_REALTIME, &requestStart);
	// Multiply A and B on GPU
	tpu_blas_mmul(handle, d_A, d_W1, d_C1, nr_rows_A, nr_cols_A, nr_cols_W1);
	hipDeviceSynchronize();
	/*clock_gettime(CLOCK_REALTIME, &requestEnd);
	timer = (double)( requestEnd.tv_sec - requestStart.tv_sec )
	+ (double)( requestEnd.tv_nsec - requestStart.tv_nsec )
	/ RES;
	std::cout << "Time Mult TPU: " << timer << std::endl;
	mult_acc += timer;

	clock_gettime(CLOCK_REALTIME, &requestStart);*/
	//Bias sum
	k_scalar_matrix_sum<<<nr_rows_A*nr_cols_W1, 1>>>(d_C1, d_bias1, d_C1);
	hipDeviceSynchronize();
	/*clock_gettime(CLOCK_REALTIME, &requestEnd);
	timer = (double)( requestEnd.tv_sec - requestStart.tv_sec )
	+ (double)( requestEnd.tv_nsec - requestStart.tv_nsec )
	/ RES;
	std::cout << "Time Bias GPU: " << timer << std::endl;
	bias_acc += timer;

	clock_gettime(CLOCK_REALTIME, &requestStart);*/
	//Bias sum
	k_relu<<<nr_rows_A*nr_cols_W1, 1>>>(d_C1, d_C1);
	hipDeviceSynchronize();
	/*clock_gettime(CLOCK_REALTIME, &requestEnd);
	timer = (double)( requestEnd.tv_sec - requestStart.tv_sec )
	+ (double)( requestEnd.tv_nsec - requestStart.tv_nsec )
	/ RES;
	std::cout << "Time Relu GPU: " << timer << std::endl;
	relu_acc += timer;

	clock_gettime(CLOCK_REALTIME, &requestStart);*/
	// Copy (and print) the result on host memory
	hipMemcpy(h_C1,d_C1,nr_rows_C1 * nr_cols_C1 * sizeof(float),hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_REALTIME, &requestEnd);
	timer = (double)( requestEnd.tv_sec - requestStart.tv_sec )
	+ (double)( requestEnd.tv_nsec - requestStart.tv_nsec )
	/ RES;
	std::cout << "Time Copy Output TPU: " << timer << std::endl;
	copy_acc+= timer;
	//std::cout << "C =" << std::endl;
	//print_matrix_float(h_C, nr_rows_C, nr_cols_C);
	//print_matrix_float(h_A, nr_rows_A, nr_cols_A);


	// Free CPU memory
	free(h_A);
	free(h_W1);
	free(h_bias1);
	free(h_C1);


	//Free GPU memory
	hipFree(d_A);
	hipFree(d_W1);
	hipFree(d_bias1);
	hipFree(d_C1);
	hipDeviceSynchronize();

	//printf("Scrivo Risultati\n");
	//fprintf(out_fp, "%1.9lf,%1.9lf,%1.9lf,%1.9lf\n" , copy_acc, mult_acc, bias_acc, relu_acc);

	//fclose(out_fp);
	// Destroy the handle
	//hipblasDestroy(handle);
}

/*
* Main Program
*/

int main(int argc, char * argv[]) {

	

	char *fullName, *weights_full, *bias_full, *fullNameRes;
	const char *pathQuery = "./Resource/lognormal/Query/", *pathWeights = "./Resource/lognormal/weights_dat/", *pathOutput = "./Result/lognormal/csv/KerasExperiment2/cudaQuery/";
	char inum[100],jnum[100];
	int file[4] = {3, 7,10, 13};
	int perc[3] = {10, 50, 80};

	for(int i = 0; i < 4; i++){
		std::cout << "Iterazione:" << i << std::endl;
		std::cout << "Apertura File" << file[i] << std::endl; 
		/*
		* Generazione file name dei pesi
		*/
		weights_full = (char*)malloc(sizeof(char)*1000);
		weights_full[0] = '\0';
		strcat(weights_full, pathWeights);
		sprintf(inum, "weights_nn1_file%d_c.dat", file[i]);
		strcat(weights_full, inum);

		/*
		* Generazione file name dei bias
		*/
		bias_full = (char*)malloc(sizeof(char)*1000);
		bias_full[0] = '\0';
		strcat(bias_full, pathWeights);
		sprintf(inum, "bias_nn1_file%d_c.dat", file[i]);
		strcat(bias_full, inum);
		
		printf("%s\n", weights_full);
		printf("%s\n", bias_full);
		if(i < 3){
			std::cout << "Sono nell'IF" << std::endl;
			for(int j = 0; j< 3; j++){
				
				/*
				* Generazione file name delle query
				*/
				fullName = (char*)malloc(sizeof(char)*1000);
				fullName[0] = '\0';
				strcat(fullName, pathQuery);
				sprintf(inum, "file%dlognormalQuery", file[i]);
				strcat(fullName, inum);
				sprintf(jnum, "%d_bin_c.dat", perc[j]);
				strcat(fullName, jnum);


				printf("%s\n", fullName);

				/*
				* Generazione file name dei risultati
				*/
				fullNameRes = (char*)malloc(sizeof(char)*1000);
				fullNameRes[0] = '\0';
				strcat(fullNameRes, pathOutput);
				sprintf(inum, "file%dlognormal", file[i]);
				strcat(fullNameRes, inum);
				sprintf(jnum, "PredTime.csv");
				strcat(fullNameRes, jnum);


				printf("%s\n", fullNameRes);

				
				prediction(fullName, weights_full, bias_full, fullNameRes);
                printf("END PREDICTION");
				/*
				*Pulisco i file dei nomi
				*/
				free(fullName);
				free(fullNameRes);
			}
		}else{
			std::cout << "Sono nell'ELSE" << std::endl;
			for(int j = 0; j< 3; j++){
				
				/*
				* Generazione file name delle query
				*/
				int flag = 1;
				int part = 1;
				while(flag){//for(int part = 1; part<=6;part++){
					fullName = (char*)malloc(sizeof(char)*1000);
					fullName[0] = '\0';
					strcat(fullName, pathQuery);
					sprintf(inum, "file%dlognormalPart%dQuery", file[i], part);
					strcat(fullName, inum);
					sprintf(jnum, "%d_bin_c.dat", perc[j]);
					strcat(fullName, jnum);


					printf("%s\n", fullName);

					/*
					* Generazione file name dei risultati
					*/
					fullNameRes = (char*)malloc(sizeof(char)*1000);
					fullNameRes[0] = '\0';
					strcat(fullNameRes, pathOutput);
					sprintf(inum, "file%dlognormalQuery", file[i]);
					strcat(fullNameRes, inum);
					sprintf(jnum, "%dPredTime.csv", perc[j]);
					strcat(fullNameRes, jnum);


					printf("%s\n", fullNameRes);

					if(fopen(fullName, "r")){
						printf("il file esiste\n");
						prediction(fullName, weights_full, bias_full, fullNameRes);
					}else{

						printf("il file non esiste\n");
						std::cout << "Errore apertura file: " << errno << std::endl;
						flag = 0;
					}

					/*
					*Pulisco i file dei nomi
					*/
					free(fullName);
					free(fullNameRes);
					part++;
				}
			}
		}

		free(weights_full);
		free(bias_full);
	}

	/*const char *fn="Resource/lognormal/Query/file3lognormalQuery10_bin_c.dat";
	const char *wfn1="Resource/lognormal/weights_dat/weights_nn1_file3_c.dat";
	const char *bfn1="Resource/lognormal/weights_dat/bias_nn1_file3_c.dat";*/
	

    return 0;
}